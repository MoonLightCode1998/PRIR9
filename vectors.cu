#include <hip/hip_runtime.h>
#include <iostream> 
using namespace std;
#include <chrono>
#include <sys/time.h>
__global__ void sumowanie(int* a, int*b, int* c){
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx<100)
    {
       c[idx]=a[idx]+b[idx];
    
    
    
    }
}

int main(){
    
  int* a_CPU=(int*)malloc(sizeof(int)*100);
  int* b_CPU=(int*)malloc(sizeof(int)*100);
  int* c_CPU=(int*)malloc(sizeof(int)*100);
  int* a_GPU;
  int* b_GPU;
  int* c_GPU;
  hipMalloc((void**)&a_GPU,sizeof(int)*100);
  hipMalloc((void**)&b_GPU,sizeof(int)*100);
  hipMalloc((void**)&c_GPU,sizeof(int)*100);
  
  for(int x =0;x<100;x++){
      a_CPU[x]=x;
      b_CPU[x]=x;
  }
  //printf("%d",a_CPU[50]);
  hipMemcpy(a_GPU,a_CPU,sizeof(int)*100,hipMemcpyHostToDevice);
  hipMemcpy(b_GPU,b_CPU,sizeof(int)*100,hipMemcpyHostToDevice);


  sumowanie<<<10,1024>>>(a_GPU,b_GPU,c_GPU);
  hipDeviceSynchronize();
  hipMemcpy(c_CPU,c_GPU,sizeof(int)*100,hipMemcpyDeviceToHost);

  printf("%d",c_CPU[1]);




  hipFree(a_GPU);
    hipFree(b_GPU);
      hipFree(c_GPU);
      free(a_CPU);
      free(b_CPU);
      free(c_CPU);






    return 0;
}