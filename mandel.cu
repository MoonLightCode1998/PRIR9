#include <hip/hip_runtime.h>
#include <iostream> 
using namespace std;
#include <chrono>
#include <sys/time.h>

__global__ void computeMandelbrot1D(double X0, double Y0, double X1, double Y1, int POZ, int PION, int ITER,int *Mandel ){
    int indx = blockIdx.x * blockDim.x + threadIdx.x;
    double Xre,Xim,Yre,Yim,Tre,Tim;
    double dy = (Y1-Y0)/PION;
    double dx=(X1-X0)/POZ;
    int k;
    
    if(indx<PION*POZ){

    
    
        Xre=(indx/POZ)*dx+X0;
            
                k=0;
                Xim=(indx%POZ)*dy+Y0;
                Yre=0;
                Yim=0;
               do{
                   Tre = Yre*Yre-Yim*Yim;
                   Tim = 2*Yre*Yim;
                   Yre=Tre;
                   Yim=Tim;
                   Yre=Yre+Xre;
                   Yim=Yim+Xim;
                   k++;
               }while(k<ITER&&(Yre*Yre+ Yim*Yim)<4);
                Mandel[indx]=k;
            
    }
    
}


void makePicture(int *Mandel,int width, int height, int MAX){
    
    int red_value, green_value, blue_value;
    
    float scale = 256.0/MAX;
    
    int MyPalette[41][3]={
        {255,255,255}, //0
        {255,255,255}, //1 not used
        {255,255,255}, //2 not used
        {255,255,255}, //3 not used
        {255,255,255}, //4 not used
        {255,180,255}, //5
        {255,180,255}, //6 not used
        {255,180,255}, //7 not used
        {248,128,240}, //8
        {248,128,240}, //9 not used
        {240,64,224}, //10
        {240,64,224}, //11 not used
        {232,32,208}, //12
        {224,16,192}, //13
        {216,8,176}, //14
        {208,4,160}, //15
        {200,2,144}, //16
        {192,1,128}, //17
        {184,0,112}, //18
        {176,0,96}, //19
        {168,0,80}, //20
        {160,0,64}, //21
        {152,0,48}, //22
        {144,0,32}, //23
        {136,0,16}, //24
        {128,0,0}, //25
        {120,16,0}, //26
        {112,32,0}, //27
        {104,48,0}, //28
        {96,64,0}, //29
        {88,80,0}, //30
        {80,96,0}, //31
        {72,112,0}, //32
        {64,128,0}, //33
        {56,144,0}, //34
        {48,160,0}, //35
        {40,176,0}, //36
        {32,192,0}, //37
        {16,224,0}, //38
        {8,240,0}, //39
        {0,0,0} //40
    };
    
    FILE *f = fopen("Mandel.ppm", "wb");
    fprintf(f, "P6\n%i %i 255\n", width, height);
    for (int j=height-1; j>=0; j--) {
        for (int i=0; i<width; i++) {
            // compute index to the palette
            int indx= (int) floor(5.0*scale*log2f(1.0f*Mandel[j*height+i]+1));
            red_value=MyPalette[indx][0];
            green_value=MyPalette[indx][2];
            blue_value=MyPalette[indx][1];
            
            fputc(red_value, f);   // 0 .. 255
            fputc(green_value, f); // 0 .. 255
            fputc(blue_value, f);  // 0 .. 255
        }
    }
    fclose(f);
    
}


int main(int argc, char **argv) {
  

    double X0=-0.82;
     double Y0=0.1; 
     double X1=-0.7;
     double Y1=0.22;
      int width=1000;
       int height=1000;
       int ITER=256;
       int threads = 1024;
       int* Mandel;
        hipMallocManaged(&Mandel,sizeof(int)*width*height);

  
    hipEvent_t start, stop;
  float elapsedTime;

  hipEventCreate(&start);
  hipEventRecord(start,0);

    computeMandelbrot1D<<<(width*height)/threads,threads>>>( X0, Y0, X1,Y1, width,height,ITER,Mandel );
    hipDeviceSynchronize();
   
 hipEventCreate(&stop);
 hipEventRecord(stop,0);
 hipEventSynchronize(stop);

 hipEventElapsedTime(&elapsedTime, start,stop);
 hipDeviceSynchronize();
 cout<<elapsedTime<<endl;

  
makePicture(Mandel,width,height,256);
hipFree(Mandel);
}
